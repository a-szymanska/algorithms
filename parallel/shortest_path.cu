#include "hip/hip_runtime.h"
/**
 * Given an undirected graph in SNAP format (https://snap.stanford.edu/data),
 * for each vertex find the maximal length of the shortest path from this vertex
 * to any vertex in the graph.
 */

#include <iostream>
#include <sstream>
#include <vector>
#include <string>
#include <algorithm>
#include "moderngpu/memory.hxx"
#include "moderngpu/transform.hxx"
#include <hip/hip_runtime.h>

__global__ void bfs_iteration(int n, int iter_cnt, int* iter, int* adj, std::pair<int, int>* adj_idx, bool* done) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) {
        return;
    }

    if (iter[id] == iter_cnt) {
        for (int i = adj_idx[id].first; i < adj_idx[id].second; i++) {
            int y = adj[i];
            if (atomicCAS(&iter[y], -1, iter_cnt + 1) == -1) {
                *done = false;
            }
        }
    }
}

int part_diameter(int x, int n, int m, int* d_adj, std::pair<int, int>* d_adj_idx) {
    std::vector<int> iter(n, -1);
    iter[x] = 0;

    int* d_iter;
    bool* d_done;

    hipMalloc(&d_iter, n * sizeof(int));
    hipMalloc(&d_done, sizeof(bool));
    hipMemcpy(d_iter, iter.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int iter_cnt = -1;
    bool done = false;
    int n_threads = 256;
    int n_blocks = (n + n_threads - 1) / n_threads;
    while (!done) {
        ++iter_cnt;
        done = true;
        hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
        bfs_iteration<<<n_blocks, n_threads>>>(n, iter_cnt, d_iter, d_adj, d_adj_idx, d_done);
        hipMemcpy(&done, d_done, sizeof(bool), hipMemcpyDeviceToHost);
    }

    hipFree(d_iter);
    hipFree(d_done);

    return iter_cnt;
}

int main() {
    std::cin.tie(nullptr);
    std::ios_base::sync_with_stdio(false);

    std::vector<std::pair<int, int>> edges;
    std::string line;
    while (std::getline(std::cin, line)) {
        if (line.empty() || line[0] == '#') {
            continue;
        }
        std::istringstream istr(line);
        int x, y;
        if (istr >> x >> y) {
            edges.emplace_back(x, y);
            edges.emplace_back(y, x);
        }
    }

    std::sort(edges.begin(), edges.end());
    int m = edges.size(), n = edges[m - 1].first + 1;
    std::vector<int> adj(m);
    std::vector<std::pair<int, int>> adj_idx(n);
    int x, y, x_prev = 0;
    adj_idx[0].first = 0;
    adj_idx[n - 1].second = m;
    for (int i = 0; i < m; i++) {
        x = edges[i].first, y = edges[i].second;
        adj[i] = y;
        if (x != x_prev) {
            adj_idx[x_prev].second = adj_idx[x].first = i;
            x_prev = x;
        }
    }

    int* d_adj;
    std::pair<int, int>* d_adj_idx;

    hipMalloc(&d_adj, m * sizeof(int));
    hipMalloc(&d_adj_idx, n * sizeof(std::pair<int, int>));
    hipMemcpy(d_adj, adj.data(), m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_adj_idx, adj_idx.data(), n * sizeof(std::pair<int, int>), hipMemcpyHostToDevice);

    for (int i = 0; i < n; i++) {
        std::cout << part_diameter(i, n, m, d_adj, d_adj_idx) << "\n";
    }

    hipFree(d_adj);
    hipFree(d_adj_idx);
    return 0;
}
